#include "hip/hip_runtime.h"
#include "hipblas.h"
#include"hip/hip_runtime_api.h"
#include<iostream>
#include<stdlib.h>
#include<stdio.h>

void tongjiliang(float*h_U, float*h_S, float *h_Tsquare, float *h_SPE, int pNcols, float*h_matrixtest, float *d_meanmatrix, float *h_biaozhuncha, int Nrows, int Ncols, int Ncolstest)
{
	float *d_U;
	float *d_S;
	float *d_t;
	float *d_t2;
	float *d_spe2;
	float *d_spe3;
	float *d_one;
	float *d_matrixtest;
	float *h_duijiaozhen = (float*)malloc(pNcols*pNcols * sizeof(float));
	float *h_one = (float*)malloc(Nrows*Nrows * sizeof(float));
	for (int i = 0; i < pNcols*pNcols; i++)
	{
		h_duijiaozhen[i] = 0;
	}
	for (int i = 0; i < pNcols; i++)
	{
		h_duijiaozhen[i*pNcols + i] = (1 / h_S[Nrows - pNcols + i]);
	}
	for (int i = 0; i <Nrows*Nrows; i++)
	{
		h_one[i] = 0;
	}
	for (int i = 0; i < Nrows; i++)
	{
		h_one[i*Nrows + i] = 1;
	}
	hipMalloc(&d_matrixtest, Nrows*Ncolstest * sizeof(float));
	hipMalloc(&d_U, Nrows*pNcols * sizeof(float));
	hipMalloc(&d_t, pNcols*Ncolstest * sizeof(float));
	hipMalloc(&d_t2, 1 * pNcols * sizeof(float));
	hipMalloc(&d_S, pNcols*pNcols * sizeof(float));
	hipMalloc(&d_one, Nrows*Nrows * sizeof(float));
	hipMalloc(&d_spe2, Nrows*Nrows * sizeof(float));
	hipMalloc(&d_spe3, 1 * Nrows * sizeof(float));
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED)
		{
			printf("CUBLAS �����ʼ������");
			getchar();
		}
	}
	hipblasSetVector(Nrows*Ncolstest, sizeof(float), h_matrixtest, 1, d_matrixtest, 1);
	hipblasSetVector(Nrows*pNcols, sizeof(float), h_U + (Nrows - pNcols)*Nrows, 1, d_U, 1);
	hipblasSetVector(pNcols*pNcols, sizeof(float), h_duijiaozhen, 1, d_S, 1);
	hipblasSetVector(Nrows*Nrows, sizeof(float), h_one, 1, d_one, 1);
	float a = 1; float b = -1;
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nrows, Ncols, &a, d_matrixtest, Nrows, &b, d_meanmatrix, Nrows, d_matrixtest, Nrows);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nrows, (Ncolstest - Ncols), &a, (d_matrixtest + Nrows*Ncols), Nrows, &b, d_meanmatrix, Nrows, (d_matrixtest + Nrows*Ncols), Nrows);
	for (int i = 0; i < Nrows; i++)
	{
		a = (sqrt(Ncols - 1)) / (*(h_biaozhuncha + i));
		hipblasSscal(handle, Ncolstest, &a, (d_matrixtest + i), Nrows);
	}
	a = 1.0, b = 0;
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, pNcols, Ncolstest, Nrows, &a, d_U, Nrows, d_matrixtest, Nrows, &b, d_t, pNcols);
	for (int i = 0; i < Ncolstest; i++)
	{
		hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, pNcols, pNcols, &a, (d_t + i*pNcols), pNcols, d_S, pNcols, &b, d_t2, 1);
		hipblasSdot(handle, pNcols, d_t2, 1, (d_t + i*pNcols), 1, (h_Tsquare + i));
	}
	hipDeviceSynchronize();
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Nrows, Nrows, pNcols, &a, d_U, Nrows, d_U, Nrows, &b, d_spe2, Nrows);
	a = 1, b = -1;
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nrows, Nrows, &a, d_one, Nrows, &b, d_spe2, Nrows, d_spe2, Nrows);
	a = 1, b = 0;
	for (int i = 0; i < Ncolstest; i++)
	{
		hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, Nrows, Nrows, &a, (d_matrixtest + i*Nrows), Nrows, d_spe2, Nrows, &b, d_spe3, 1);
		hipblasSdot(handle, Nrows, d_spe3, 1, (d_matrixtest + i*Nrows), 1, (h_SPE + i));
	}
	hipDeviceSynchronize();
	hipblasDestroy(handle);
	free(h_duijiaozhen);
	free(h_one);
	hipFree(d_matrixtest);
	hipFree(d_U);
	hipFree(d_t);
	hipFree(d_t2);
	hipFree(d_S);
	hipFree(d_one);
	hipFree(d_spe2);
	hipFree(d_spe3);
}
