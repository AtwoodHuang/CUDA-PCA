#include "hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
#include "hipblas.h"
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
float* covarianceMatrix(float*h_matrix, float *d_meanmatrix, float *h_biaozhuncha, int Nrows, int Ncols)
{
	float *d_matrix;
	float *d_covarianceMatrix;
	float * h_Ncols = (float *)malloc(Ncols * Ncols * sizeof(float));;
	float *d_Ncols;
	for (int i = 0; i < Ncols*Ncols; i++)
		h_Ncols[i] = 1.0 / Ncols;
	hipMalloc(&d_matrix, Nrows*Ncols * sizeof(float));
	hipMalloc(&d_Ncols, Ncols*Ncols * sizeof(float));
	hipMalloc(&d_covarianceMatrix, Nrows*Nrows * sizeof(float));
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED)
		{
			printf("CUBLAS �����ʼ������");
			getchar();
		}
	}
	hipblasSetVector(Nrows*Ncols, sizeof(float), h_matrix, 1, d_matrix, 1);
	hipblasSetVector(Ncols*Ncols, sizeof(float), h_Ncols, 1, d_Ncols, 1);
	float a = 1; float b = 0;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nrows, Ncols, Ncols, &a, d_matrix, Nrows, d_Ncols, Ncols, &b, d_meanmatrix, Nrows);
	a = 1; b = -1;
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nrows, Ncols, &a, d_matrix, Nrows, &b, d_meanmatrix, Nrows, d_matrix, Nrows);
	for (int i = 0; i < Nrows; i++)
		hipblasSnrm2(handle, Ncols, (d_matrix + i), Nrows, (h_biaozhuncha + i));
	hipDeviceSynchronize();
	for (int i = 0; i < Nrows; i++)
	{
		a = (sqrt(Ncols - 1)) / (*(h_biaozhuncha + i));
		hipblasSscal(handle, Ncols, &a, (d_matrix + i), Nrows);
	}
	a = 1.0 / (Ncols - 1); b = 0;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Nrows, Nrows, Ncols, &a, d_matrix, Nrows, d_matrix, Nrows, &b, d_covarianceMatrix, Nrows);
	hipblasDestroy(handle);
	free(h_Ncols);
	hipFree(d_matrix);
	hipFree(d_Ncols);
	return d_covarianceMatrix;

}