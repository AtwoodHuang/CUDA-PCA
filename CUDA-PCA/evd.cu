#include "hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
#include<stdio.h>
#include<stdlib.h>
#include <hipsolver.h>
#include "Utilities.cuh"
void evd(float* d_covarianceMatrix, float*h_U, float*h_S, int Nrows, int Ncols)
{
	// cusolverǰ�ڲ���
	int work_size = 0;
	int *devInfo;           gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

	// cusolver��ʼ��
	hipsolverHandle_t solver_handle;
	hipsolverDnCreate(&solver_handle);

	// �ԽǾ���
	float *d_S;            gpuErrchk(hipMalloc(&d_S, Nrows * sizeof(float)));

	// ��������ֵ�ֽ�����ռ�
	cusolveSafeCall(hipsolverDnSsyevd_bufferSize(solver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, Nrows, d_covarianceMatrix, Nrows, d_S, &work_size));
	float *work;   gpuErrchk(hipMalloc(&work, work_size * sizeof(float)));

	// ���ú�������ֵ�ֽ�
	cusolveSafeCall(hipsolverDnSsyevd(solver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, Nrows, d_covarianceMatrix, Nrows, d_S, work, work_size, devInfo));
	int devInfo_h = 0;  gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	if (devInfo_h != 0) printf("Unsuccessful SVD execution\n\n");

	// ��GPUȡ������
	gpuErrchk(hipMemcpy(h_S, d_S, Nrows * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_U, d_covarianceMatrix, Nrows * Nrows * sizeof(float), hipMemcpyDeviceToHost));//��������������ԭ��ꇿ��g
																										  //�ͷſռ�
	hipFree(devInfo);
	hipFree(work);
	hipFree(d_S);
	hipsolverDnDestroy(solver_handle);
}