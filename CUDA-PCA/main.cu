#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include"hip/hip_runtime_api.h"
#include <time.h>
#include <windows.h>
float* covarianceMatrix(float*h_matrix, float *d_meanmatrix, float *h_biaozhuncha, int Nrows, int Ncols);
void evd(float* d_covarianceMatrix, float*h_U, float*h_S, int Nrows, int Ncols);
void Matrixinput(float*h_matrix, float*h_matrixtest, int Nrows, int Ncols, int Nrowstest, int Ncolstest, bool zhengchang, bool ceshi, char **argv);
void tongjiliang(float*h_U, float*h_S, float *h_Tsquare, float *h_SPE, int pNcols, float*h_matrixtest, float *d_meanmatrix, float *h_biaozhuncha, int Nrows, int Ncols, int Ncolstest);
int main(int argc, char **argv)
{
	clock_t startime;
	clock_t endtime;
	startime = clock();
	bool zhengchang = true;
	bool ceshi = true;
	int Nrows = atoi(argv[3]);
	int Ncols = atoi(argv[4]);
	int Nrowstest = atoi(argv[5]);
	int Ncolstest = atoi(argv[6]);
	if (Nrows > Ncols)
	{
		zhengchang = false;
		int c = Nrows;
		Nrows = Ncols;
		Ncols = c;
	}
	if (Nrowstest > Ncolstest)
	{
		ceshi = false;
		int d = Nrowstest;
		Nrowstest = Ncolstest;
		Ncolstest = d;
	}
	float sum = 0;
	float sum1 = 0;
	int pNcols = 0;
	float *h_U = (float *)malloc(Nrows * Nrows * sizeof(float));
	float *h_S = (float *)malloc(Nrows * sizeof(float));
	float *d_covarianceMatrix;
	float *d_meanmatrix;
	float *h_biaozhuncha = (float*)malloc(sizeof(float)*Nrows);
	float *h_matrix = (float *)malloc(Nrows * Ncols * sizeof(float));
	float *h_matrixtest = (float *)malloc(Nrows * Ncolstest * sizeof(float));
	float *h_Tsquare = (float *)malloc(Ncolstest * sizeof(float));
	float *h_SPE = (float *)malloc(Ncolstest * sizeof(float));
	hipMalloc(&d_meanmatrix, Nrows*Ncols * sizeof(float));
	Matrixinput(h_matrix, h_matrixtest, Nrows, Ncols, Nrowstest, Ncolstest, zhengchang, ceshi, argv);
	d_covarianceMatrix = covarianceMatrix(h_matrix, d_meanmatrix, h_biaozhuncha, Nrows, Ncols);
	evd(d_covarianceMatrix, h_U, h_S, Nrows, Ncols);
	for (int i = 0; i < Nrows; i++)
	{
		sum = sum + h_S[i];
	}
	for (int i = 0; i < Nrows; i++)
	{
		sum1 = sum1 + h_S[Nrows - 1 - i];
		pNcols++;
		if ((sum1 / sum) > 0.85)
			break;
	}
	tongjiliang(h_U, h_S, h_Tsquare, h_SPE, pNcols, h_matrixtest, d_meanmatrix, h_biaozhuncha, Nrows, Ncols, Ncolstest);
	printf("��Ԫ����p=%d\n", pNcols);
	printf("�Խ���\n");
	for (int i = 0; i < pNcols; i++)
		printf("S[%d]=%e\n", i, h_S[Nrows - 1 - i]);
	printf("��ԪP\n");
	for (int i = 0; i < Nrows; i++)
	{
		for (int j = 0; j < pNcols; j++)
		{
			printf("P[%d,%d]=%e ", i, j, h_U[(Nrows - 1 - j)*Nrows + i]);
			if (j + 1 == pNcols)
				printf("\n");
		}
	}
	FILE *fp3 = fopen("SPE.txt", "w");
	for (int i = 0; i < Ncolstest; i++)
	{
		fprintf(fp3, "%e\n", h_SPE[i]);
	}
	FILE *fp4 = fopen("Tsquare.txt", "w");
	for (int i = 0; i < Ncolstest; i++)
	{
		fprintf(fp4, "%e\n", h_Tsquare[i]);
	}
	fclose(fp3);
	fclose(fp4);
	free(h_U);
	free(h_S);
	free(h_matrix);
	free(h_matrixtest);
	free(h_Tsquare);
	free(h_SPE);
	free(h_biaozhuncha);
	hipFree(d_meanmatrix);
	endtime = clock();
	printf("��������ʱ�䣺%dms\n", endtime - startime);
	getchar();
	return 0;
}